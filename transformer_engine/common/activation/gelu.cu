#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/activation.h>
#include <hip/hip_runtime.h>
#include <cfloat>
#include <iostream>
#include "../utils.cuh"
#include "../common.h"
#include <cstdlib>
#include <../util/vectorized_pointwise.h>

namespace transformer_engine {

namespace detail {

struct GELUParam {
  const fp32 *scale_inv;
};

__device__ inline fp32 gelu(fp32 value, const GELUParam &) {
  return value * (0.5F + 0.5F * tanhf(value * (0.79788456F + 0.03567741F * value * value)));
}
__device__ inline fp32 gelu_dequantize(fp32 value, const GELUParam &p) {
  value = value * (*(p.scale_inv));
  return value * (0.5F + 0.5F * tanhf(value * (0.79788456F + 0.03567741F * value * value)));
}

}

void gelu_cast(const Tensor &input,
               Tensor *output,
               hipStream_t stream) {
  CheckInputTensor(input, "gelu_input");
  CheckOutputTensor(*output, "gelu_output");
  NVTE_CHECK(input.data.shape.size() == 2, "Input must have 2 dimensions.");
  NVTE_CHECK(output->data.shape.size() == 2, "Output must have 2 dimensions.");
  NVTE_CHECK(input.data.shape == output->data.shape, "Input and output shapes must match.");
  const size_t tot_elts = input.data.shape[1] * input.data.shape[0];

  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(input.data.dtype, IType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(output->data.dtype, OType,
      constexpr int nvec = 32 / sizeof(IType);
      detail::GELUParam p;
      p.scale_inv = reinterpret_cast<fp32*>(input.scale_inv.dptr);
      VectorizedUnaryKernelLauncher<nvec, detail::GELUParam, detail::gelu>(
        reinterpret_cast<const IType*>(input.data.dptr),
        reinterpret_cast<OType*>(output->data.dptr),
        reinterpret_cast<const fp32*>(output->scale.dptr),
        reinterpret_cast<fp32*>(output->scale_inv.dptr),
        reinterpret_cast<fp32*>(output->amax.dptr),
        tot_elts,
	p,
        stream);
    );  // NOLINT(*)
  );  // NOLINT(*)
}

constexpr int gelu_kernel_threads = 512;

template <int nvec, bool aligned,
          typename ComputeType,
          typename Param,
          typename InputType,
          typename OutputType>
__launch_bounds__(gelu_kernel_threads)
__global__ void fp8_gelu_kernel(const InputType *input,
                             OutputType *output,
                             const ComputeType *scale,
                             ComputeType *scale_inv,
                             ComputeType *amax,
                             Param p,
                             const size_t N,
                             const size_t num_aligned_elements) {
  VectorizedLoader<InputType, nvec, aligned> loader(input, N);
  VectorizedStorer<OutputType, nvec, aligned> storer(output, N);
  ComputeType max = 0;
  ComputeType s = 0;
  if (scale != nullptr) s = *scale;
  if (blockIdx.x == 0 && threadIdx.x == 0 && scale_inv != nullptr) {
    reciprocal<ComputeType>(scale_inv, s);
  }
  const int warp_id = threadIdx.x / THREADS_PER_WARP;

  const size_t M = num_aligned_elements;

  for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
       tid < M;
       tid += gridDim.x * blockDim.x) {
    loader.load(tid, N);
#pragma unroll
    for (int i = 0; i < nvec; ++i) {
      const ComputeType val = static_cast<ComputeType>(loader.separate()[i]) * (*(p.scale_inv));
      // ComputeType temp = OP(val, p);
      ComputeType t0 = 0.79788456F + 0.03567741F * val * val;
      ComputeType tanh = tanhf(val * t0);
      ComputeType temp = val * (0.5F + 0.5F * tanh);
      __builtin_assume(max >= 0);
      max = fmaxf(fabsf(temp), max);

      temp = temp * s;

      storer.separate()[i] = static_cast<OutputType>(temp);
    }
    storer.store(tid, N);
  }
  /* warp tile amax reduce*/
  max = reduce_max<gelu_kernel_threads / THREADS_PER_WARP>(max, warp_id);

  if (threadIdx.x == 0 && amax != nullptr) {
      static_assert(std::is_same<ComputeType, float>::value);
      atomicMaxFloat(amax, max);
  }
}

template <int nvec, typename Param,
          typename InputType,
          typename OutputType>
void fp8_gelu_fp8input(const InputType *input,
                       OutputType *output,
                       const fp32 *scale,
                       fp32 *scale_inv,
                       fp32 *amax,
                       const size_t N,
                       const Param params,
                       hipStream_t stream) {
  if (N != 0) {
    auto align = CheckAlignment(N, nvec, input, output);

    size_t num_aligned_elements = get_num_aligned_elements(input, N, nvec,
                                                           sizeof(InputType));
    constexpr size_t threads = gelu_kernel_threads;
    size_t num_blocks = DIVUP(num_aligned_elements, threads);
    constexpr size_t max_blocks = 65535;
    num_blocks = std::min(num_blocks, max_blocks);

    switch (align) {
      case Alignment::SAME_ALIGNED:
        fp8_gelu_kernel<nvec, true, fp32, Param><<<num_blocks, threads, 0, stream>>>(
            input, output, scale, scale_inv, amax, params, N, num_aligned_elements);
        break;
      case Alignment::SAME_UNALIGNED:
        fp8_gelu_kernel<nvec, false, fp32, Param><<<num_blocks, threads, 0, stream>>>(
            input, output, scale, scale_inv, amax, params, N, num_aligned_elements);
        break;
      case Alignment::DIFFERENT: {
        // If the pointers are aligned differently we cannot vectorize
        fp8_gelu_kernel<1, true, fp32, Param><<<num_blocks, threads, 0, stream>>>(
            input, output, scale, scale_inv, amax, params, N, N);
        break;
      }
    }
  }
}

void gelu_cast_fp8input(const Tensor &input,
               Tensor *output,
               hipStream_t stream) {
  CheckInputTensor(input, "gelu_input");
  CheckOutputTensor(*output, "gelu_output");
  NVTE_CHECK(input.data.shape.size() == 2, "Input must have 2 dimensions.");
  NVTE_CHECK(output->data.shape.size() == 2, "Output must have 2 dimensions.");
  NVTE_CHECK(input.data.shape == output->data.shape, "Input and output shapes must match.");
  const size_t tot_elts = input.data.shape[1] * input.data.shape[0];

  TRANSFORMER_ENGINE_TYPE_SWITCH_FP8ONLY(input.data.dtype, IType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_FP8ONLY(output->data.dtype, OType,
      constexpr int nvec = 32 / sizeof(IType);
      detail::GELUParam p;
      p.scale_inv = reinterpret_cast<fp32*>(input.scale_inv.dptr);
      fp8_gelu_fp8input<nvec, detail::GELUParam>(
        reinterpret_cast<const IType*>(input.data.dptr),
        reinterpret_cast<OType*>(output->data.dptr),
        reinterpret_cast<const fp32*>(output->scale.dptr),
        reinterpret_cast<fp32*>(output->scale_inv.dptr),
        reinterpret_cast<fp32*>(output->amax.dptr),
        tot_elts,
        p,
        stream);
    );  // NOLINT(*)
  );  // NOLINT(*)
}

}  // namespace transformer_engine

void nvte_gelu(const NVTETensor input,
               NVTETensor output,
               hipStream_t stream) {
  using namespace transformer_engine;
  gelu_cast(*reinterpret_cast<const Tensor*>(input),
            reinterpret_cast<Tensor*>(output),
            stream);
}
void nvte_gelu_fp8input(const NVTETensor input,
               NVTETensor output,
               hipStream_t stream) {
  using namespace transformer_engine;
  gelu_cast_fp8input(*reinterpret_cast<const Tensor*>(input),
            reinterpret_cast<Tensor*>(output),
            stream);
}
